#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>
#include <map>
#include <string>
#include <vector>

#include "closure.cuh"
#include "globals.h"
#include "utils.h"

using namespace std;

struct State {
    bool s[NUM_USERS * NUM_ROLES];
};

__global__ void analysis_kernel(bool *states, // current existing states
                                bool *worksetIn, bool *worksetOut,
                                int *worksetIdx, int *worksetSize, bool *s,
                                bool *relCanAssign, bool *relPos, bool *relNeg,
                                int numUsers, int numRules, int numRoles) {
    int user = threadIdx.x + 1;
    int rule = threadIdx.y;
    int stateIndex = blockIdx.x; // This thread deals with worksetIn[stateIndex]
    int blockSize = blockDim.x * blockDim.y * blockDim.z;

    int blockStateCounter[blockSize]; // need to be locked

    // should be defined in main function
    bool worksetOut[blockSize * numRoles * numUsers * 10];

    if (user >= numUsers || rule >= numRules) {
        return;
    }

    __shared__ bool;

    __shared__ int cond2Flag[NUM_USERS * NUM_CA_RULES];
    __shared__ int cond3Flag[NUM_USERS * NUM_CA_RULES];
    __shared__ int cond4Flag[NUM_USERS * NUM_CA_RULES];

    if (threadIdx.z == 0)
        cond2Flag[user * numRules + rule] = 0;
    if (threadIdx.z == 0)
        cond3Flag[user * numRules + rule] = 0;
    if (threadIdx.z == 0)
        cond4Flag[user * numRules + rule] = 0;

    __syncthreads();

    int role = threadIdx.z;

    bool cond1 = false;
    bool cond2 = false;
    bool cond3 = false;
    bool cond4 = false;

    // Cond1 and Cond2
    if (role < numRoles) {
        cond1 = (relCanAssign[numRoles * 4 * rule + numRoles * 3 + role] && (relPos[role] && relNeg[role]));
    }

    // Cond2
    if (role < numRoles && (!relCanAssign[numRoles * 4 * rule + numRoles + role] || s[user * numRoles + role]) == 0) {
        atomicExch(&cond2Flag[user * numRules + rule], 1);
    }

    // Cond3
    if (role < numRoles && (s[user * numRoles + role] && relCanAssign[numRoles * 4 * rule + numRoles * 2 + role])) {
        atomicExch(&cond3Flag[user * numRules + rule], 1);
    }

    // Cond4
    if (role < numRoles && s[role] && relCanAssign[numRoles * 4 * rule + role]) {
        atomicExch(&cond4Flag[user * numRules + rule], 1);
    }

    __syncthreads();

    if (cond2Flag[user * numRules + rule] == 1)
        cond2 = false;
    if (cond3Flag[user * numRules + rule] == 1)
        cond3 = false;
    if (cond4Flag[user * numRules + rule] == 1)
        cond4 = true;

    bool allCond = cond1 && cond2 && cond3 && cond4;

    if (allCond) {
        // Get closure(s + (user, role))
        // Check if state is reached
        // Add closure(state) to the worksetOut
        s[role * numRoles + role] = s[user * numRoles + role] || allCond;
        s[role] = s[role] || allCond;
        int offset = (5 * blockDim.x + blockStateCounter[blockDim.x] * numUsers * numRoles);
        for (int i = 0; i < numUsers; i++) {
            for (int j = 0; j < numRoles; j++) {
                worksetOut[offset + i * j + j] = s[i * j + j];
            }
        }
        blockStateCounter[blockDim.x]++;
    }

    __syncthreads();
    // return worksetOut;
}

int main() {
    bool CA[NUM_CA_RULES][4][NUM_ROLES] = {{{1, 0, 0, 0, 0, 0, 0, 0},
                                            {0, 1, 0, 0, 0, 0, 0, 0},
                                            {0, 0, 0, 0, 0, 0, 0, 0},
                                            {0, 0, 1, 0, 0, 0, 0, 0}},
                                           {{0, 0, 0, 0, 0, 1, 0, 0},
                                            {0, 0, 1, 1, 0, 0, 0, 0},
                                            {0, 0, 0, 0, 0, 0, 0, 0},
                                            {0, 0, 0, 0, 1, 0, 0, 0}},
                                           {{1, 0, 0, 0, 0, 0, 0, 0},
                                            {0, 0, 0, 0, 0, 1, 0, 0},
                                            {0, 0, 1, 0, 0, 0, 0, 0},
                                            {0, 0, 0, 1, 0, 0, 0, 0}},
                                           {{0, 1, 0, 0, 0, 0, 0, 0},
                                            {1, 0, 0, 0, 0, 0, 0, 1},
                                            {0, 0, 0, 0, 0, 0, 0, 0},
                                            {0, 0, 0, 0, 0, 1, 0, 0}},
                                           {{0, 1, 0, 0, 0, 0, 0, 0},
                                            {0, 0, 0, 0, 0, 1, 0, 0},
                                            {0, 0, 0, 0, 0, 0, 0, 0},
                                            {0, 0, 0, 0, 0, 0, 1, 0}}};

    bool s[NUM_USERS][NUM_ROLES] = {{1, 1, 1, 0, 0, 1, 0, 1},
                                    {1, 0, 1, 0, 0, 0, 0, 0},
                                    {0, 1, 0, 0, 0, 0, 0, 1},
                                    {0, 1, 0, 0, 0, 0, 0, 1},
                                    {0, 0, 0, 0, 0, 1, 0, 0}};

    bool relPos[NUM_ROLES] = {1, 1, 1, 1, 1, 1, 0, 1};
    bool relNeg[NUM_ROLES] = {0, 0, 1, 0, 0, 0, 0, 0};

    bool worksetIn[MAX_STATES_WORKSET][NUM_USERS][NUM_ROLES] = {};
    // bool worksetOut[MAX_STATES_WORKSET * 5][NUM_USERS][NUM_ROLES] = {};
    bool worksetOut[MAX_STATES_WORKSET * 5 * NUM_USERS * NUM_ROLES];

    // loadWorkset(worksetIn, s);
    loadWorkset(&worksetIn[0][0][0], &s[0][0]);

    vector<bool *> States;

    bool *d_relPos;
    bool *d_relNeg;
    bool *d_s;
    bool *d_CA;
    bool *d_worksetIn;
    bool *d_worksetOut;

    hipMalloc(&d_relPos, NUM_ROLES * sizeof(bool));
    hipMalloc(&d_relNeg, NUM_ROLES * sizeof(bool));

    hipMalloc(&d_s, NUM_USERS * NUM_ROLES * sizeof(bool));
    hipMalloc((void **)&d_CA, NUM_CA_RULES * 4 * NUM_ROLES * sizeof(bool));

    hipMemcpy(d_relPos, relPos, NUM_ROLES * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_relNeg, relNeg, NUM_ROLES * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_s, s, NUM_USERS * NUM_ROLES * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_CA, CA, NUM_CA_RULES * 4 * NUM_ROLES * sizeof(bool), hipMemcpyHostToDevice);

    int blockSize = 1;

    dim3 gridDim(blockSize);
    dim3 blockDim(NUM_USERS, NUM_CA_RULES, NUM_ROLES);

    std::string stateAscii = getStateAscii(&s[0][0]);
    printf("State Ascii:\n%s\nlength: %d\n", stateAscii, stateAscii.length());

    // closure_kernel<<<gridDim, blockDim>>>(d_s, d_CA, d_relPos, d_relNeg, NUM_USERS, NUM_CA_RULES, NUM_ROLES);
    // TODO: flatten all arrays
    map<string, int> stateIdMap;
    vector<State> pendingStates;
    int idCounter = 0;

    while (!pendingStates.empty()) {

        // TODO: rewrite closure_kernel to return closure states
        // worksetOut = closure_kernel<<<gridDim, blockDim>>>(d_s, d_CA, d_relPos, d_relNeg, NUM_USERS, NUM_CA_RULES, NUM_ROLES);
        for (int i = 0; i < 5 * MAX_STATES_WORKSET; i++) {
            // if ascii not in map then add <ascii, id> into map
            State currentState;
            for (int j = 0; j < NUM_USERS; j++) {
                for (int k = 0; k < NUM_ROLES; k++) {
                    currentState.s[j * NUM_ROLES + k] = worksetOut[i * NUM_USERS * NUM_ROLES + j * NUM_ROLES + k];
                }
            }
            string asciiState = getStateAscii(currentState.s);
            if (stateIdMap.find(asciiState) == stateIdMap.end()) {
                stateIdMap.insert({asciiState, idCounter++});
            }
            pendingStates.push_back(currentState);
        }
        for (int i = 0; i < blockSize; i++) {
            // assign new worksetIn
        }
        // worksetOut = analysis(worksetIn)
    }

    /*
    while (!workSet.empty()) {
        analysis_kernel<<<gridDim, blockDim>>>(d_s, d_CA, d_relPos, d_relNeg,
    NUM_USERS, NUM_CA_RULES, NUM_ROLES);

    }
    */

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        cout << "CUDA kernel launch failed: " << hipGetErrorString(err) << endl;
        return -1;
    }

    hipDeviceSynchronize();

    hipMemcpy(s, d_s, NUM_USERS * NUM_ROLES * sizeof(bool), hipMemcpyDeviceToHost);

    for (int i = 0; i < NUM_USERS; i++) {
        for (int j = 0; j < NUM_ROLES; j++) {
            cout << s[i][j] << " ";
        }
        cout << endl;
    }

    hipFree(d_relPos);
    hipFree(d_relNeg);
    hipFree(d_s);
    hipFree(d_CA);

    return 0;
}
