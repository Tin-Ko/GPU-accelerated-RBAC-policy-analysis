#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>
#include <map>
#include <string>
#include <vector>

#include "analysis.cuh"
#include "closure.cuh"
#include "globals.h"

using namespace std;

__global__ void analysis_kernel(bool *worksetIn, bool *worksetOut, int worksetOutIndex,
                                bool *relCanAssign, bool *relPos, bool *relNeg,
                                int numUsers, int numRules, int numRoles) {
    int user = threadIdx.x + 1;
    int rule = threadIdx.y;
    int stateIndex = blockIdx.x; // This thread deals with worksetIn[stateIndex]
    int blockSize = blockDim.x * blockDim.y * blockDim.z;

    int blockStateCounter[blockSize]; // need to be locked

    int stateSize = numUsers * numRoles;

    // should be defined in main function

    if (user >= numUsers || rule >= numRules) {
        return;
    }

    __shared__ int cond2Flag[NUM_USERS * NUM_CA_RULES];
    __shared__ int cond3Flag[NUM_USERS * NUM_CA_RULES];
    __shared__ int cond4Flag[NUM_USERS * NUM_CA_RULES];

    if (threadIdx.z == 0)
        cond2Flag[user * numRules + rule] = 0;
    if (threadIdx.z == 0)
        cond3Flag[user * numRules + rule] = 0;
    if (threadIdx.z == 0)
        cond4Flag[user * numRules + rule] = 0;

    __syncthreads();

    int role = threadIdx.z;

    bool cond1 = false;
    bool cond2 = false;
    bool cond3 = false;
    bool cond4 = false;

    // Cond1 and Cond2
    if (role < numRoles) {
        cond1 = (relCanAssign[numRoles * 4 * rule + numRoles * 3 + role] && (relPos[role] && relNeg[role]));
    }

    // Cond2
    if (role < numRoles && (!relCanAssign[numRoles * 4 * rule + numRoles + role] || worksetIn[stateIndex * stateSize + user * numRoles + role]) == 0) {
        atomicExch(&cond2Flag[user * numRules + rule], 1);
    }

    // Cond3
    if (role < numRoles && (worksetIn[stateIndex * stateSize + user * numRoles + role] && relCanAssign[numRoles * 4 * rule + numRoles * 2 + role])) {
        atomicExch(&cond3Flag[user * numRules + rule], 1);
    }

    // Cond4
    if (role < numRoles && worksetIn[stateIndex * stateSize + role] && relCanAssign[numRoles * 4 * rule + role]) {
        atomicExch(&cond4Flag[user * numRules + rule], 1);
    }

    __syncthreads();

    if (cond2Flag[user * numRules + rule] == 1)
        cond2 = false;
    if (cond3Flag[user * numRules + rule] == 1)
        cond3 = false;
    if (cond4Flag[user * numRules + rule] == 1)
        cond4 = true;

    bool allCond = cond1 && cond2 && cond3 && cond4;

    __syncthreads();

    if (allCond) {
        bool newState[numUsers * numRoles] = worksetIn[stateIndex * stateSize + user * numRoles + role];
        worksetOut[worksetOutIndex * stateSize + user * numRoles + role] = worksetIn[stateIndex * stateSize + user * numRoles + role] || allCond;
        worksetOut[worksetOutIndex * stateSize + role] = worksetIn[stateIndex * stateSize + role] || allCond;
        // int offset = (5 * blockDim.x + blockStateCounter[blockDim.x] * numUsers * numRoles);
        // blockStateCounter[blockDim.x]++;
    }

    __syncthreads();
}