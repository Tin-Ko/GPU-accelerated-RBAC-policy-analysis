#include <iostream>
#include <hip/hip_runtime.h>


#define NUM_USERS 5
#define NUM_ROLES 8
#define NUM_CA_RULES 5


__global__ void closure_kernel(bool* s, bool* relCanAssign, bool* relPos, bool* relNeg, int numUsers, int numRules, int numRoles) {
    // int tid = threadIdx.x;
    int user = threadIdx.x + 1;   // Starts from user 1
    int rule = threadIdx.y;

    if (user >= numUsers || rule >= numRules) {
        return;
    }

    __shared__ int cond2Flag[NUM_USERS * NUM_CA_RULES];
    __shared__ int cond3Flag[NUM_USERS * NUM_CA_RULES];
    __shared__ int cond4Flag[NUM_USERS * NUM_CA_RULES];

    if (threadIdx.z == 0) cond2Flag[user * numRules + rule] = 0;
    if (threadIdx.z == 0) cond3Flag[user * numRules + rule] = 0;
    if (threadIdx.z == 0) cond4Flag[user * numRules + rule] = 0;

    __syncthreads();

    int role = threadIdx.z;

    bool cond1 = false;
    bool cond2 = true;
    bool cond3 = true;
    bool cond4 = false;

    // Cond1 and Cond2
    if (role < numRoles) {
        cond1 = (relCanAssign[numRoles * 4 * rule + numRoles * 3  + role] && (relPos[role] && !relNeg[role]));
    }
    
    // Cond2
    if (role < numRoles && (!relCanAssign[numRoles * 4 * rule + numRoles + role] || s[user * numRoles + role]) == 0) {
        atomicExch(&cond2Flag[user * numRules + rule], 1);
    }


    // Cond3
    if (role < numRoles && (s[user * numRoles + role] && relCanAssign[numRoles * 4 * rule + numRoles * 2 + role])) {
        atomicExch(&cond3Flag[user * numRules + rule], 1);
    }

    
    // Cond4
    if (role < numRoles && s[role] && relCanAssign[numRoles * 4 * rule + role]) {
        atomicExch(&cond4Flag[user * numRules + rule], 1);
    }

    __syncthreads();

    if (cond2Flag[user * numRules + rule] == 1) cond2 = false;
    if (cond3Flag[user * numRules + rule] == 1) cond3 = false;
    if (cond4Flag[user * numRules + rule] == 1) cond4 = true;


    bool allCond = cond1 && cond2 && cond3 && cond4;

    if (allCond) {
        s[user * numRoles + role] = s[user * numRoles + role] || allCond;
        s[role] = s[role] || allCond;
    }
    
    __syncthreads();

}
