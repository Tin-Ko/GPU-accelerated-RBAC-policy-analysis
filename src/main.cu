#include <hip/hip_runtime.h>

#include <iostream>
#include <map>
#include <string>
#include <vector>

#include "analysis.cuh"
#include "closure.cuh"
#include "globals.h"
#include "utils.h"

using namespace std;

struct State {
    bool s[NUM_USERS * NUM_ROLES];
};

int main() {
    bool CA[NUM_CA_RULES * 4 * NUM_ROLES] = {
        // rule 0
        1, 0, 0, 0, 0, 0, 0, 0,
        0, 1, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 1, 0, 0, 0, 0, 0,
        // rule 1
        0, 0, 0, 0, 0, 1, 0, 0,
        0, 0, 1, 1, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 1, 0, 0, 0,
        // rule 2
        1, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 1, 0, 0,
        0, 0, 1, 0, 0, 0, 0, 0,
        0, 0, 0, 1, 0, 0, 0, 0,
        // rule 3
        0, 1, 0, 0, 0, 0, 0, 0,
        1, 0, 0, 0, 0, 0, 0, 1,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 1, 0, 0,
        // rule 4
        0, 1, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 1, 0, 0,
        0, 0, 0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0, 1, 0};

    bool s[NUM_USERS * NUM_ROLES] = {1, 1, 1, 0, 0, 1, 0, 1,
                                     1, 0, 1, 0, 0, 0, 0, 0,
                                     0, 1, 0, 0, 0, 0, 0, 1,
                                     0, 1, 0, 0, 0, 0, 0, 1,
                                     0, 0, 0, 0, 0, 1, 0, 0};

    bool relPos[NUM_ROLES] = {1, 1, 1, 1, 1, 1, 0, 1};
    bool relNeg[NUM_ROLES] = {0, 0, 1, 0, 0, 0, 0, 0};

    bool worksetIn[MAX_STATES_WORKSET * NUM_USERS * NUM_ROLES] = {};
    // bool worksetOut[MAX_STATES_WORKSET * 5][NUM_USERS][NUM_ROLES] = {};
    bool worksetOut[MAX_STATES_WORKSET * 5 * NUM_USERS * NUM_ROLES];
    int worksetOutIndex = -1;

    int goalUser = 4;
    int goalRole = 5;

    int goalReached = 1;

    loadWorkset(worksetIn, s, 0);

    vector<bool *> States;

    bool *d_relPos;
    bool *d_relNeg;
    bool *d_s;
    bool *d_CA;
    bool *d_worksetIn;
    bool *d_worksetOut;

    int *d_worksetOutIndex;

    int *d_goalReached;

    hipMalloc(&d_relPos, NUM_ROLES * sizeof(bool));
    hipMalloc(&d_relNeg, NUM_ROLES * sizeof(bool));

    hipMalloc(&d_s, NUM_USERS * NUM_ROLES * sizeof(bool));
    hipMalloc(&d_CA, NUM_CA_RULES * 4 * NUM_ROLES * sizeof(bool));

    hipMalloc(&d_worksetIn, MAX_STATES_WORKSET * NUM_USERS * NUM_ROLES * sizeof(bool));
    hipMalloc(&d_worksetOut, MAX_STATES_WORKSET * 5 * NUM_USERS * NUM_ROLES * sizeof(bool));
    hipMalloc(&d_worksetOutIndex, sizeof(int));

    hipMalloc(&d_goalReached, sizeof(bool));

    hipMemcpy(d_relPos, relPos, NUM_ROLES * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_relNeg, relNeg, NUM_ROLES * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_s, s, NUM_USERS * NUM_ROLES * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_CA, CA, NUM_CA_RULES * 4 * NUM_ROLES * sizeof(bool), hipMemcpyHostToDevice);

    hipMemcpy(d_worksetOutIndex, &worksetOutIndex, sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_goalReached, &goalReached, sizeof(int), hipMemcpyHostToDevice);

    dim3 gridDim(MAX_STATES_WORKSET);
    dim3 blockDim(NUM_USERS, NUM_CA_RULES, NUM_ROLES);

    map<string, int> stateIdMap;
    vector<State> pendingStates;
    int idCounter = 0;

    // for (int n = 0; n < MAX_STATES_WORKSET; ++n) {
    //     for (int i = 0; i < NUM_USERS; i++) {
    //         for (int j = 0; j < NUM_ROLES; j++) {
    //             cout << worksetIn[n * NUM_USERS * NUM_ROLES + i * NUM_ROLES + j] << " ";
    //         }
    //         cout << endl;
    //     }
    // }

    while (!pendingStates.empty()) {
        hipMemcpy(d_worksetOut, worksetOut, MAX_STATES_WORKSET * 5 * NUM_USERS * NUM_ROLES * sizeof(bool), hipMemcpyHostToDevice);
        closure_kernel<<<gridDim, blockDim>>>(d_worksetOut, d_CA, d_relPos, d_relNeg, NUM_USERS, NUM_CA_RULES, NUM_ROLES, goalUser, goalRole, d_goalReached);
        hipMemcpy(worksetOut, d_worksetOut, MAX_STATES_WORKSET * 5 * NUM_USERS * NUM_ROLES * sizeof(bool), hipMemcpyDeviceToHost);

        for (int i = 0; i < 5 * MAX_STATES_WORKSET; i++) {
            // if ascii not in map then add <ascii, id> into map
            State currentState;
            for (int j = 0; j < NUM_USERS; j++) {
                for (int k = 0; k < NUM_ROLES; k++) {
                    currentState.s[j * NUM_ROLES + k] = worksetOut[i * NUM_USERS * NUM_ROLES + j * NUM_ROLES + k];
                }
            }
            string asciiState = getStateAscii(currentState.s);
            if (stateIdMap.find(asciiState) == stateIdMap.end()) {
                stateIdMap.insert({asciiState, idCounter++});
                pendingStates.push_back(currentState);
            }
        }
        for (int i = 0; i < MAX_STATES_WORKSET; i++) {
            // Assign new worksetIn
            for (int j = 0; j < NUM_USERS * NUM_ROLES; ++j) {
                worksetIn[i * NUM_USERS * NUM_ROLES + j] = pendingStates[i].s[j];
            }
            pendingStates.erase(pendingStates.begin());
        }
        hipMemcpy(d_worksetIn, worksetIn, MAX_STATES_WORKSET * NUM_USERS * NUM_ROLES * sizeof(bool), hipMemcpyHostToDevice);
        analysis_kernel<<<gridDim, blockDim>>>(d_worksetIn, d_worksetOut, d_worksetOutIndex, d_CA, d_relPos, d_relNeg, NUM_USERS, NUM_CA_RULES, NUM_ROLES, goalUser, goalRole, d_goalReached);
        hipMemcpy(worksetOut, d_worksetOut, MAX_STATES_WORKSET * 5 * NUM_USERS * NUM_ROLES * sizeof(bool), hipMemcpyDeviceToHost);
    }

    // closure_kernel<<<gridDim, blockDim>>>(d_s, d_CA, d_relPos, d_relNeg, NUM_USERS, NUM_CA_RULES, NUM_ROLES);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        cout << "CUDA kernel launch failed: " << hipGetErrorString(err) << endl;
        return -1;
    }

    hipDeviceSynchronize();

    hipMemcpy(&goalReached, d_goalReached, sizeof(int), hipMemcpyDeviceToHost);

    printf("Goal Reached: %d", goalReached);

    hipMemcpy(s, d_s, NUM_USERS * NUM_ROLES * sizeof(bool), hipMemcpyDeviceToHost);

    // for (int i = 0; i < NUM_USERS; i++) {
    //     for (int j = 0; j < NUM_ROLES; j++) {
    //         cout << s[i * NUM_ROLES + j] << " ";
    //     }
    //     cout << endl;
    // }

    hipFree(d_relPos);
    hipFree(d_relNeg);
    hipFree(d_s);
    hipFree(d_CA);
    hipFree(d_worksetIn);
    hipFree(d_worksetOut);
    hipFree(d_worksetOutIndex);

    return 0;
}
